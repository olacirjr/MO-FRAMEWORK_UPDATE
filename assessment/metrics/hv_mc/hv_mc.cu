#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <values.h>
#include <string.h>
#include <time.h>

const int maxObjectiveNumber=20;
int objectiveNumber;
//const int tamanhoAmostra=1000000;
const int tamanhoAmostra=1000;


const int maxSize=10000;
double dados[maxSize][maxObjectiveNumber];
__device__ double d_dados[maxSize][maxObjectiveNumber];
__device__ int d_contaAcertos[maxSize];
int contaAcertos[maxSize];

double refPoint;
int tamRef;
hiprandState* devStates; //devstates of the hiprand generator

int blockSize;
int nBlocks;

int lerArquivos(char* arquivo);
int main(int argc, char* argv[]){
	if(argc < 3){
		printf("\nuse: hv <file> <ref_point>");
	}
	hipMalloc ( &devStates, maxSize*sizeof( hiprandState ) );
	
	refPoint=atof(argv[2]);
	lerArquivos(argv[1]);
}

//function used in the file reading
void parse( char *record, char *delim, char arr[][1024],int *fldcnt){
    char*p=strtok(record,delim);
    int fld=0;
    
    while(p != NULL)
    {
        strcpy(arr[fld],p);
		fld++;
		p=strtok('\0',delim);
	}		
	*fldcnt=fld;
}

__global__ void calculaHipervolumeAmostragem(hiprandState* devStates, int objectiveNumber, int tamRef, double refPoint, int cont, int it);
//function to read the files an throw they values to the matrices in the memory
int lerArquivos(char* arquivo){
	int cont=0;
	char tmp[4096];
	int fldcnt=0;
	char arr[1000][1024];
	int recordcnt=0;
	FILE *in=fopen(arquivo,"r");         // open file on command line 
	
	if(in==NULL)
	{
		perror("Error opening the file\n");
		exit(EXIT_FAILURE);
	}
	while(fgets(tmp,sizeof(tmp),in)!=0){ // read a record 
		parse(tmp,(char*)" \t",arr,&fldcnt);   // whack record into fields
		if ((fldcnt != 1 || (!strcmp(arr[0]," \n") && strcmp(arr[0],"#\n") ) ) ){
			
			for(int coluna=0;coluna<fldcnt;coluna++){
// 				for(int i=0;i<strlen(arr[coluna]);i++){
// 					if(arr[coluna][i] == ',')
// 						arr[coluna][i]='.';
// 				}
				dados[recordcnt][coluna]=(double)atof(arr[coluna]);
			}
			recordcnt++;
			objectiveNumber=fldcnt-1;
		}else{
			if(recordcnt > 0){
				tamRef=recordcnt;
				if(tamRef > maxSize){
					printf("\nERROR! Front size is larger than the maximum allowed! (%d)\n", tamRef);
					exit(1);
				}
				
				//blockSize = 8;
				//blockSize=64;
				blockSize=256;
				
				
				nBlocks = tamanhoAmostra/blockSize + (tamanhoAmostra%blockSize == 0?0:1);
				hipMemcpyToSymbol(HIP_SYMBOL(d_dados), &dados, sizeof(double)*maxSize*maxObjectiveNumber, 0, hipMemcpyHostToDevice);
				int tamMul=1000;
				for(int i=0;i<tamMul;i++){
					calculaHipervolumeAmostragem <<< nBlocks, blockSize >>> (devStates, objectiveNumber, tamRef, refPoint, cont, i);
					hipDeviceSynchronize();
				}
				hipMemcpyFromSymbol(&contaAcertos, HIP_SYMBOL(d_contaAcertos), sizeof(int)*tamanhoAmostra, 0, hipMemcpyDeviceToHost);
				int total=0;
				for(int i=0;i<tamanhoAmostra;i++)
					total+=contaAcertos[i];
				
				double volumeTotal = pow(refPoint, objectiveNumber);
				double hv = (double) total / (double) tamanhoAmostra * volumeTotal;
				hv/=tamMul;
				cont++;
				printf("hv(%d) = %.10f\n", cont, hv);
				
				const char* lastError=hipGetErrorString(hipGetLastError());
				
				if(strcmp(lastError, "no error")){
					printf("\nCuda error status: %s\n", lastError);
					exit(1);
				}
				
// 				printf("\nac: %d hv: %f",total, hv/tamMul);
// 				printf("\n val: %s ", arr[0]);
// 				printf("rec: %d, fld: %d\n", recordcnt, objectiveNumber);
				recordcnt=0;
				//break;
			}
		}
	}	
	fclose(in);
	//objectiveNumber=fldcnt;
	return recordcnt;
}

__global__ void calculaHipervolumeAmostragem(hiprandState* devStates, int objectiveNumber, int tamRef, double refPoint, int cont, int it){
// 	for(int i=0;i<tamRef;i++){
// 		for(int j=0;j<objectiveNumber;j++)
// 			printf("%f " , dados[i][j]);
// 		printf("\n");
// 	}
// 	return 0;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx<tamanhoAmostra){
		if(cont==0 && it ==0)
			hiprand_init ( clock(), idx, 0, &devStates[idx] );
		
		//hiprandState localState = devStates[idx];
		//int contaAcertos = 0;
		if(it==0)
			d_contaAcertos[idx]=0;
		__shared__ double gerado[maxObjectiveNumber];
		// Modificar a linha abaixo caso os dados não estejam normalizados
		// ou estejam normalizados em um intervalo de tamanho diferente
		// 1.1 = 2.1 - 1.0, ou seja, limite superior - limite inferior
		
		//no meu caso 1.0 = 1 - 0
		
		// Se os dados tiverem diferentes limites para os diferentes objetivos
		// calcular o volumeTotal pelo produtório das diferenças dos limites....
		//double volumeTotal = pow(refPoint, objectiveNumber);
		bool dominado = true;


		//for (int i = 0; i < tamanhoAmostra; i++) {
			for (int j = 0; j < objectiveNumber; j++) {
				//Gera cada dimensao de um ponto...
				//gerado[j] = 1.0 + 1.1 * random.nextDouble();
				//gerado[j] = refPoint * rand()/RAND_MAX;
				gerado[j] = refPoint * (double)hiprand_uniform(&devStates[idx]);
				//limites uniformes...
				//gerado[j] = limiteInferior + (limiteSuperior - limiteInferior) * random.nextDouble();
				//diferente limites para cada dimensao...
				//gerado[j] = limiteInferior[j] + (limiteSuperior[j] - limiteInferior[j]) * random.nextDouble();
// 				printf("%f ",gerado[j]);
			}
// 			printf(" (%f)\n", refPoint);

			dominado = false;

			//Verifica se o ponto é dominado ou não...
			for (int k = 0; ((k < tamRef) && (!dominado)); k++) {
				double* temp = d_dados[k];
				
				bool dominadoTemp = true;

				for (int d = 0; d < objectiveNumber; d++) {
					if (temp[d] > gerado[d]) {
						dominadoTemp = false;
					}
// 					printf("%f ",temp[d]);
				}
// 				printf("\n");

				if (dominadoTemp) {
					dominado = true;
				}
			}

			if (dominado) {
				d_contaAcertos[idx]++;//conta o número de pontos dominados...
			}

			//if(i%1000 == 0)
			//System.out.println(i);
		//}
		//devStates[idx] = localState;
		//System.out.println(contaAcertos);

		//O hipervolume é proporcional a quantidade de pontos dominados e ao volume da área amostrada...
		//return (double) contaAcertos / (double) tamanhoAmostra * volumeTotal;
	}

}

